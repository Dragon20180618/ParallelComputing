
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void print(){
    printf("hello from the GPU");
}
int main(){
    print<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
//在就算能力达到3.0之前，CUDA并未实现通过GPU直接printf打印的功能