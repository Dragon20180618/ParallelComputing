
#include <hip/hip_runtime.h>
#include<stdio.h>
void __global__ add(const double *d_x,const double *d_y,double *d_z)
{
    const int n =blockDim.x*blockIdx.x+threadIdx.x;
    d_z[n]=d_x[n]+d_y[n];
}
int main(){
    double *d_x,*d_y,*d_z;
    double *h_x,*h_y,*h_z;
    hipMalloc((void **)&d_x, 128*8);
    hipMalloc((void **)&d_y, 128*8);
    hipMalloc((void **)&d_z, 128*8);
    h_x=(double *)malloc(128*8);
    h_y=(double *)malloc(128*8);
    h_z=(double *)malloc(128*8);
    for(int i=0;i<128;i++)
    {
        h_x[i]=1.23;
        h_y[i]=2.34;
    }
    hipMemcpy(d_x,h_x,128*8,hipMemcpyHostToDevice);
    hipMemcpy(d_y,h_y,128*8,hipMemcpyHostToDevice);
    add<<<2,64>>>(d_x,d_y,d_z);
    hipDeviceSynchronize();
    hipMemcpy(h_z,d_z,128*8,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i = 1 ; i<128;i++)
    {
        h_z[0]+=h_z[i];
    }
    printf("%lf\n",h_z[0]);
    return 0;
}
