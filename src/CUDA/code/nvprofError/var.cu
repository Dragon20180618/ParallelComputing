#define  _CRT_SECURE_NO_WARNINGS

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel(int *a){
    int id=threadIdx.x;
    a[id]+=a[id];
}
extern "C" void cudafunction_(){
    hipSetDevice(3);
    int *a,*b,r=0;
    a=(int*)malloc(sizeof(int)*1000);
    hipMalloc((void **)&b,sizeof(int)*1000);
    for(int i=0;i<1000;i++){
        a[i]=1;
    }
    hipMemcpy(b,a,sizeof(int)*1000,hipMemcpyHostToDevice);
    kernel<<<1,1000>>>(b);
    hipDeviceSynchronize();
    hipMemcpy(a,b,sizeof(int)*1000,hipMemcpyDeviceToHost);
    for(int i=0;i<1000;i++){
        r+=a[i];
    }
    printf("Final result is : %d\n",r);
}