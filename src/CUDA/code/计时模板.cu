hipEvent_t start, stop;
CHECK(hipEventCreate(&start));
CHECK(hipEventCreate(&stop));
CHECK(hipEventRecord(start));//Record 记录
hipEventQuery(start);//Can't Check
/*
    需要计时的代码块
*/
CHECK(hipEventRecord(stop));
CHECK(hipEventSynchronize(stop));
float elapsed_time; //elapesd 过去的  adj.
CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
printf("Time = %g ms.\n",elapsed_time);
CHECK(cudaEventDestory(start));
CHECK(cudaEventDestory(stop));
