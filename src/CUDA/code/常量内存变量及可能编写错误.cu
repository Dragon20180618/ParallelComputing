
#include <hip/hip_runtime.h>
#include<stdio.h>
__constant__ int c_a[3];
//int c_a[3] error
//__constant__ int c_a[] error
//__constant__ int c_a[3]={1,2,3} error
void __global__ print(){
    for(int i = 0;i<3;i++){
        printf("%d ",c_a[i]);
    }
}
int main(){
    int a[]={1,2,3};
    //cudaMemcpy(c_a,a,12,cudaMemcpyHostToDevice) error
    hipMemcpyToSymbol(HIP_SYMBOL(c_a),a,12,0,hipMemcpyHostToDevice);
    print<<<1,1>>>();
}
/*cudaMemcpyToSymbol
cudaError_t cudaMemcpyToSymbol	(	
    const void * 	symbol, 常量内存地址
    const void * 	src,   数据来源地址
    size_t 	count,      拷贝的字节数
    size_t 	offset,     偏移量，如果是0，则从src起始位置开始
    enum cudaMemcpyKind 	kind	传递类型，填写cudaMemcpyHostToDevices 
    )	
*/